#include <iostream>
#include <hip/hip_runtime.h>

__global__ void matmul(float *a, float *b, float *c, int N)
{
    int row = threadIdx.y, col = threadIdx.x;
    if (row < N && col < N)
    {
        float val = 0;
        for (int i = 0; i < N; ++i)
            val += a[row * N + i] * b[i * N + col];
        c[row * N + col] = val;
    }
}

int main()
{
    const int N = 3, SIZE = N * N * sizeof(float);
    float a[9] = {1, 2, 3, 4, 5, 6, 7, 8, 9}, b[9] = {9, 8, 7, 6, 5, 4, 3, 2, 1}, c[9];

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, SIZE);
    hipMalloc(&d_b, SIZE);
    hipMalloc(&d_c, SIZE);
    hipMemcpy(d_a, a, SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, SIZE, hipMemcpyHostToDevice);

    matmul<<<1, dim3(N, N)>>>(d_a, d_b, d_c, N);
    hipMemcpy(c, d_c, SIZE, hipMemcpyDeviceToHost);

    for (int i = 0; i < 9; ++i)
    {
        std::cout << c[i] << (i % N == N - 1 ? "\n" : " ");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
